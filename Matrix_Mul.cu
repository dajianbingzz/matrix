#include <iostream>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
using namespace std;
#define BLOCK_WIDTH 16
#define BLOCK_HIGH 16
#include <stdio.h>
//朴素矩阵乘法
__global__ void Mat_Mul(float *A,float*B,float*C,int WIDTH,int ComLong)
{
    //获得行、列
    int row=BLOCK_HIGH*blockIdx.y+threadIdx.y;
    int col=BLOCK_WIDTH*blockIdx.x+threadIdx.x;
    float Cvalue=0;
    //A中每一行的每一个元素*B中每一列的每一个元素的和
    for(int k=0;k<ComLong;k++)
    {
        Cvalue+=A[row*ComLong+k]*B[k*WIDTH+col];
    }
    C[row*WIDTH+col]=Cvalue;
}

//使用共享内存的矩阵乘法
__global__ void Mat_Mul_shard(float *A,float*B,float*C,int WIDTH,int ComLong)
{
    //定义共享内存大小
    __shared__ float Ads[BLOCK_WIDTH][BLOCK_WIDTH];
    __shared__ float Bds[BLOCK_WIDTH][BLOCK_WIDTH];
    //获得行、列
    int row=BLOCK_HIGH*blockIdx.y+threadIdx.y;
    int col=BLOCK_WIDTH*blockIdx.x+threadIdx.x;
    float Cvalue=0;
    //
    for(int m=0;m<(ComLong+BLOCK_WIDTH-1)/BLOCK_WIDTH;m++)
    {
        //把数据存入共享内存
        Ads[threadIdx.y][threadIdx.x]=A[row*ComLong+(m*BLOCK_WIDTH+threadIdx.x)];
        Bds[threadIdx.y][threadIdx.x]=B[(m*BLOCK_WIDTH+threadIdx.y)*WIDTH+col];
        __syncthreads();
        //计算当前共享block里的行*列
        for(int k=0;k<BLOCK_WIDTH;k++)
        {
        Cvalue+=Ads[threadIdx.y][k]*Bds[k][threadIdx.x];
        }
         __syncthreads();//这里不同步可能就有线程把后续block里的数据存到shared上，导致取的数据是错的 
    }
    C[row*WIDTH+col]=Cvalue;    
}

//使用有bank冲突的共享内存的矩阵乘法
__global__ void Mat_Mul_shard_bank(float *A,float*B,float*C,int WIDTH,int ComLong)
{
    //定义共享内存大小
    __shared__ float Ads[BLOCK_WIDTH][BLOCK_WIDTH];
    __shared__ float Bds[BLOCK_WIDTH][BLOCK_WIDTH];
    //获得行、列
    int row=BLOCK_HIGH*blockIdx.x+threadIdx.x;
    int col=BLOCK_WIDTH*blockIdx.y+threadIdx.y;
    float Cvalue=0;
    //
    for(int m=0;m<(ComLong+BLOCK_WIDTH-1)/BLOCK_WIDTH;m++)
    {
        //把数据存入共享内存
        Ads[threadIdx.x][threadIdx.y]=A[row*ComLong+(m*BLOCK_WIDTH+threadIdx.y)];
        Bds[threadIdx.x][threadIdx.y]=B[(m*BLOCK_WIDTH+threadIdx.x)*WIDTH+col];
        __syncthreads();
        //计算当前共享block里的行*列
        for(int k=0;k<BLOCK_WIDTH;k++)
        {
        Cvalue+=Ads[threadIdx.x][k]*Bds[k][threadIdx.y];
        }
         __syncthreads();//这里不同步可能就有线程把后续block里的数据存到shared上，导致取的数据是错的 
    }
    C[row*WIDTH+col]=Cvalue;    
}

// //test使用有bank冲突的共享内存的矩阵乘法
// __global__ void Mat_Mul_shard_bank(float *A,float*B,float*C,int WIDTH,int ComLong)
// {
//     //定义共享内存大小
//     __shared__ float Ads[BLOCK_WIDTH][BLOCK_WIDTH];
//     __shared__ float Bds[BLOCK_WIDTH][BLOCK_WIDTH];
//     //获得行、列
//     int row=BLOCK_HIGH*blockIdx.y+threadIdx.y;
//     int col=BLOCK_WIDTH*blockIdx.x+threadIdx.x;
//     float Cvalue=0;
//     //
//     for(int m=0;m<(ComLong+BLOCK_WIDTH-1)/BLOCK_WIDTH;m++)
//     {
//         //把数据存入共享内存
//         Ads[threadIdx.y][threadIdx.x]=A[row*ComLong+(m*BLOCK_WIDTH+threadIdx.x)];
//         Bds[threadIdx.y][threadIdx.x]=B[(m*BLOCK_WIDTH+threadIdx.y)*WIDTH+col];
//         __syncthreads();
//         //计算当前共享block里的行*列
//         for(int k=0;k<BLOCK_WIDTH;k++)
//         {
//         Cvalue+=Ads[threadIdx.y][k]*Bds[k][threadIdx.x];
//         }
//          __syncthreads();//这里不同步可能就有线程把后续block里的数据存到shared上，导致取的数据是错的 
//     }
//     C[row*WIDTH+col]=Cvalue;    
// }
// //解决了bank冲突的共享内存的矩阵乘法
__global__ void Mat_Mul_shard_bank_fix(float *A,float*B,float*C,int WIDTH,int ComLong)
{
    //定义共享内存大小
    __shared__ float Ads[BLOCK_WIDTH][BLOCK_WIDTH+1];
    __shared__ float Bds[BLOCK_WIDTH][BLOCK_WIDTH+1];
    //获得行、列
    int row=BLOCK_HIGH*blockIdx.x+threadIdx.x;
    int col=BLOCK_WIDTH*blockIdx.y+threadIdx.y;
    float Cvalue=0;
    //
    for(int m=0;m<(ComLong+BLOCK_WIDTH-1)/BLOCK_WIDTH;m++)
    {
        //把数据存入共享内存
        Ads[threadIdx.x][threadIdx.y]=A[row*ComLong+(m*BLOCK_WIDTH+threadIdx.y)];
        Bds[threadIdx.x][threadIdx.y]=B[(m*BLOCK_WIDTH+threadIdx.x)*WIDTH+col];
        __syncthreads();
        //计算当前共享block里的行*列
        for(int k=0;k<BLOCK_WIDTH;k++)
        {
        Cvalue+=Ads[threadIdx.x][k]*Bds[k][threadIdx.y];
        }
         __syncthreads();//这里不同步可能就有线程把后续block里的数据存到shared上，导致取的数据是错的 
    }
    C[row*WIDTH+col]=Cvalue;    
}
int main()
{
    const int WIDTH=3096;
    const int HIGH=3096;
    const int Xab=3096;
    //printf("WIDTH=%d\n",WIDTH); 
   // 定义cpu上的矩阵
    float (*array1_h)[Xab] = (float (*)[Xab])malloc(HIGH * sizeof(float[Xab]));
    // float *array1_h = malloc(WIDTH * sizeof(float[WIDTH]));
    // float (*array1_h)[B] = (float (*)[B])malloc(A * sizeof(float[B]));
    // array1_h[B][A]
    float (*array2_h)[WIDTH] = (float (*)[WIDTH])malloc(Xab* sizeof(float[WIDTH]));
    float (*result_h)[WIDTH] = (float(*)[WIDTH])malloc(HIGH*sizeof(float[WIDTH]));
    float (*test_h)[WIDTH] = (float(*)[WIDTH])malloc(HIGH*sizeof(float[WIDTH]));//测试矩阵
      //初始化矩阵
    int i=0,j=0;
    for(i=0;i<HIGH;i++)//array1_h初始化
    {
        for(j=0;j<Xab;j++)
        {
            array1_h[i][j]=i;
        }
    }
     for(i=0;i<Xab;i++)//array2_h初始化
    {
        for(j=0;j<WIDTH;j++)
        {
            array2_h[i][j]=j;
        }
    }
    //printf("array1_h=%f\n",array1_h[1][10]);
    //printf("array2_h=%f\n",array2_h[10][1]);
    //cpu运算矩阵乘法
    for(i=0;i<HIGH;i++)
    {
        for(j=0;j<WIDTH;j++)
        {
            for(int k=0;k<Xab;k++)
            {
            test_h[i][j]+=array1_h[i][k]*array2_h[k][j];
            }
        }
    }
    //printf("test_h[0][1]=%f\n",test_h[1][1]);
    // printf("test_h[1][1]=%f\n",test_h[1][1]);
    // printf("test_h[2][1]=%f\n",test_h[2][1]);
    // printf("test_h[2][2]=%f\n",test_h[2][2]);
    //定义时间事件
    hipEvent_t start,stop1,stop2;
    float elapsedTime,elapsedTimecpy,timeall;
    hipEventCreate(&start);
    hipEventCreate(&stop1);
    hipEventCreate(&stop2);
    //定义GPU上矩阵
    int size=WIDTH*HIGH;
    int bytes=size*sizeof(int);
    //printf("size=%d,bytes=%d\n",size,bytes);
    float *array1_d;
    float *array2_d;
    float *result_d;
    hipMalloc((void**)&array1_d,HIGH*Xab*sizeof(float));
    hipMalloc((void**)&array2_d,Xab*WIDTH*sizeof(float));
    hipMalloc((void**)&result_d,bytes);
    

    //cpu传GPU
    hipEventRecord(start);
    hipMemcpy(array1_d,array1_h,HIGH*Xab*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(array2_d,array2_h,Xab*WIDTH*sizeof(float),hipMemcpyHostToDevice);
    
    //定义kernel函数的执行设置
    dim3 blocksize(BLOCK_WIDTH,BLOCK_HIGH,1);
    dim3 gridsize((WIDTH+BLOCK_WIDTH-1)/BLOCK_WIDTH,(WIDTH+BLOCK_HIGH-1)/BLOCK_HIGH,1);
    printf("blocksize.x=%d,blocksize.y=%d,blocksize.z=%d\n",blocksize.x,blocksize.y,blocksize.z);
    printf("gridsize.x=%d,gridsize.y=%d,gridsize.z=%d\n",gridsize.x,gridsize.y,gridsize.z);
    hipEventRecord(stop1);
    hipEventElapsedTime(&elapsedTimecpy,start,stop1);
    printf("cpy time=%d\n",elapsedTimecpy);
   //热身
    Mat_Mul<<<gridsize,blocksize>>>(array1_d,array2_d,result_d,WIDTH,Xab);
   // 执行kernel函数

    //朴素矩阵乘法
    hipEventRecord(start,0);
    Mat_Mul<<<gridsize,blocksize>>>(array1_d,array2_d,result_d,WIDTH,Xab);
    
        //GPU->CPU
    hipMemcpy(result_h,result_d,bytes,hipMemcpyDeviceToHost);
    hipEventRecord(stop1,0);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&elapsedTime,start,stop1);
    timeall=elapsedTime+elapsedTimecpy;
    printf("Mat_MUl runtime=%f,Mat_MUl alltime=%f\n",elapsedTime,timeall);
        //验证
    // printf("result_h[1][1]%f\n",result_h[1][1]);
    // printf("result_h[2][1]%f\n",result_h[2][1]);
    // printf("result_h[2][2]%f\n",result_h[2][2]);
    for(i=0;i<WIDTH;i++)
    {
        
        for(j=0;j<WIDTH;j++)
        {
            if(fabs(test_h[i][j]-result_h[i][j])!=0)
            {
                printf("Result verification failed at element%d\n",i*WIDTH+j);
                exit(EXIT_FAILURE);
            }
        }
    }
    printf("testpass\n");

   //使用了共享内存的矩阵乘法
    hipEventRecord(start,0);
    Mat_Mul_shard<<<gridsize,blocksize>>>(array1_d,array2_d,result_d,WIDTH,Xab);
        //GPU->CPU
    hipMemcpy(result_h,result_d,bytes,hipMemcpyDeviceToHost);
    hipEventRecord(stop2,0);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&elapsedTime,start,stop2);
    timeall=elapsedTime+elapsedTimecpy;
    printf("Mat_MUl_shared runtime=%f,Mat_MUl_shared alltime=%f\n",elapsedTime,timeall);
        //验证
    // printf("result_h[1][1]%f\n",result_h[1][1]);
    // printf("result_h[2][1]%f\n",result_h[2][1]);
    // printf("result_h[2][2]%f\n",result_h[2][2]);
    for(i=0;i<WIDTH;i++)
    {
        for(j=0;j<WIDTH;j++)
        {
            if(fabs(test_h[i][j]-result_h[i][j])!=0)
            {
                printf("Result verification failed at element%d\n",i*WIDTH+j);
                exit(EXIT_FAILURE);
            }
        }
    }
    printf("testpass\n");

     // //使用bank冲突的共享内存的矩阵乘法
    hipEventRecord(start,0);
    Mat_Mul_shard_bank<<<gridsize,blocksize>>>(array1_d,array2_d,result_d,WIDTH,Xab);
        //GPU->CPU
    hipMemcpy(result_h,result_d,bytes,hipMemcpyDeviceToHost);
    hipEventRecord(stop2,0);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&elapsedTime,start,stop2);
    timeall=elapsedTime+elapsedTimecpy;
    printf("Mat_MUl_shared_bank runtime=%f,Mat_MUl_shared_bank alltime=%f\n",elapsedTime,timeall);
    // //   验证
    // printf("result_h[1][1]%f\n",result_h[1][1]);
    // printf("result_h[2][1]%f\n",result_h[2][1]);
    // printf("result_h[2][2]%f\n",result_h[2][2]);
    for(i=0;i<WIDTH;i++)
    {
        for(j=0;j<WIDTH;j++)
        {
            if(fabs(test_h[i][j]-result_h[i][j])!=0)
            {
                printf("Result verification failed at element%d\n",i*WIDTH+j);
                exit(EXIT_FAILURE);
            }
        }
    }
    printf("testpass\n");

    //解决bank冲突的共享内存的矩阵乘法
    hipEventRecord(start,0);
    Mat_Mul_shard_bank_fix<<<gridsize,blocksize>>>(array1_d,array2_d,result_d,WIDTH,Xab);
            //GPU->CPU
    hipMemcpy(result_h,result_d,bytes,hipMemcpyDeviceToHost);
    hipEventRecord(stop2,0);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&elapsedTime,start,stop2);
    timeall=elapsedTime+elapsedTimecpy;
    printf("Mat_MUl_shared_bank_fix runtime=%f,Mat_MUl_shared_bank_fix alltime=%f\n",elapsedTime,timeall);
    // //验证
    // printf("result_h[1][1]%f\n",result_h[1][1]);
    // printf("result_h[2][1]%f\n",result_h[2][1]);
    // printf("result_h[2][2]%f\n",result_h[2][2]);
    for(i=0;i<WIDTH;i++)
    {
        for(j=0;j<WIDTH;j++)
        {
            if(fabs(test_h[i][j]-result_h[i][j])!=0)
            {
                printf("Result verification failed at element%d\n",i*WIDTH+j);
                exit(EXIT_FAILURE);
            }
        }
    }
    printf("testpass\n");
    //使用共享内存+流的矩阵乘法
    //  //为了使用流申请的
    // float *array1_d1;
    // float *array2_d1;
    // float *result_d1;
    // float *array1_d2;
    // float *array2_d2;
    // float *result_d2;
    // cudaHostAlloc((void**)&array1_d1,HIGH*Xab*sizeof(float)/2,cudaHostAllocDefault);
    // cudaHostAlloc((void**)&array2_d1,Xab*WIDTH*sizeof(float)/2,cudaHostAllocDefault);
    // cudaHostAlloc((void**)&result_d1,bytes/2,cudaHostAllocDefault);
    // cudaHostAlloc((void**)&array1_d2,HIGH*Xab*sizeof(float)/2,cudaHostAllocDefault);
    // cudaHostAlloc((void**)&array2_d2,Xab*WIDTH*sizeof(float)/2,cudaHostAllocDefault);
    // cudaHostAlloc((void**)&result_d2,bytes/2,cudaHostAllocDefault);
    // cudaStream_t stream0,stream1;
    // cudaStreamCreate(&stream0);
    // cudaStreamCreate(&stream1);
    // //计算开始
    // printf("1\n");
    // //cudaEventRecord(start,0);
    // cudaMemcpyAsync(array1_d1,array1_h,HIGH*Xab*sizeof(float)/2,cudaMemcpyHostToDevice,stream0);
    // cudaMemcpyAsync(array1_d2,array1_h+256,HIGH*Xab*sizeof(float)/2,cudaMemcpyHostToDevice,stream1);
    // cudaMemcpyAsync(array2_d1,array2_h,Xab*WIDTH*sizeof(float)/2,cudaMemcpyHostToDevice,stream0);
    // //cudaMemcpyAsync(array2_d2,array2_h+(Xab*WIDTH)/2,Xab*WIDTH*sizeof(float)/2,cudaMemcpyHostToDevice,stream1);
    // dim3 blocksize_1(BLOCK_WIDTH,BLOCK_HIGH,1);
    // dim3 gridsize_1(((WIDTH+BLOCK_WIDTH-1)/BLOCK_WIDTH)/2,((WIDTH+BLOCK_HIGH-1)/BLOCK_HIGH,1)/2);
    // printf("2\n");
    // Mat_Mul_shard<<<gridsize_1,blocksize_1,0,stream0>>>(array1_d1,array2_d1,result_d1,WIDTH,Xab);
    // //Mat_Mul_shard<<<gridsize_1,blocksize_1,0,stream1>>>(array1_d2,array2_d2,result_d2,WIDTH,Xab);
    // printf("3\n");
    // cudaMemcpyAsync(result_h,result_d1,bytes/2,cudaMemcpyDeviceToHost,stream0);
    // //cudaMemcpyAsync(result_h+WIDTH*HIGH/2,result_d2,bytes/2,cudaMemcpyDeviceToHost,stream1);
    // printf("4\n");
    //  // //验证
    // // printf("result_h[1][1]%f\n",result_h[1][1]);
    // // printf("result_h[2][1]%f\n",result_h[2][1]);
    // // printf("result_h[2][2]%f\n",result_h[2][2]);
    // for(i=0;i<WIDTH;i++)
    // {
    //     for(j=0;j<WIDTH;j++)
    //     {
    //         if(fabs(test_h[i][j]-result_h[i][j])!=0)
    //         {
    //             printf("Result verification failed at element%d\n",i*WIDTH+j);
    //             exit(EXIT_FAILURE);
    //         }
    //     }
    // }
    // printf("testpass\n");
    // cudaStreamSynchronize(stream0);
    // cudaStreamSynchronize(stream1);
    // //释放空间
    // cudaFree(array1_d);
    // cudaFree(array2_d);
    // cudaFree(result_d);
    // cudaFreeHost(array1_h);
    // cudaFreeHost(array2_h);
    // cudaFreeHost(result_h);
    
    // cudaFreeHost(array1_d1);
    // cudaFreeHost(array2_d1);
    // cudaFreeHost(result_d1);
    // cudaFreeHost(array1_d2);
    // cudaFreeHost(array2_d2);
    // cudaFreeHost(result_d2);
    // cudaStreamDestroy(stream0);
    // cudaStreamDestroy(stream1);
   
    return 0;
}