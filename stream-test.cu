#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <math.h>  
#include <hip/hip_runtime.h>
using namespace std;
#define N (1024*1024)//每个流的大小
#define FULL (N*20)//全部数据大小
__global__ void kernel_add(int *a,int *b,int*c)
{
    int idx=blockDim.x*blockIdx.x+threadIdx.x;
    //int idy=blockdim.y*blockId.y+threadId.y;
    c[idx]=a[idx]+b[idx];
}
int main()
{
    //查询设备属性
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop,whichDevice);
    if(!prop.deviceOverlap)
    {
        printf("Device will not support\n");
        return 0;
    }
    else 
    {printf("device pass\n");}

    //初始化计时器事件
    hipEvent_t start,stop;
    float elapsedTime;
    //申明流和buffer指针
    hipStream_t stream0,stream1,stream2,stream3;
    int *a_d,*b_d,*c_d;
    int *a_h,*b_h,*c_h;
        
    int *a1_d,*b1_d,*c1_d;
    int *a2_d,*b2_d,*c2_d;
    int *a3_d,*b3_d,*c3_d;
    //申请GPU空间
    hipMalloc((void**)&a_d,N*sizeof(int));
    hipMalloc((void**)&b_d,N*sizeof(int));
    hipMalloc((void**)&c_d,N*sizeof(int));
    
    hipMalloc((void**)&a1_d,N*sizeof(int));
    hipMalloc((void**)&b1_d,N*sizeof(int));
    hipMalloc((void**)&c1_d,N*sizeof(int));

    hipMalloc((void**)&a2_d,N*sizeof(int));
    hipMalloc((void**)&b2_d,N*sizeof(int));
    hipMalloc((void**)&c2_d,N*sizeof(int));

    hipMalloc((void**)&a3_d,N*sizeof(int));
    hipMalloc((void**)&b3_d,N*sizeof(int));
    hipMalloc((void**)&c3_d,N*sizeof(int));
    //在cpu端申请内存空间，用锁页内存
    hipHostAlloc((void**)&a_h,FULL*sizeof(int),hipHostMallocDefault);
    hipHostAlloc((void**)&b_h,FULL*sizeof(int),hipHostMallocDefault);
    hipHostAlloc((void**)&c_h,FULL*sizeof(int),hipHostMallocDefault);
    
    //创建时间寄存器
    hipEventCreate(&start);
    hipEventCreate(&stop);
   
    //初始化流
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);   
    //初始化A，B向量
    for(int i=0;i<FULL;i++)
    {
        a_h[i]=i;
        b_h[i]=2*i;
    }
  
    //开始计算1个流的方法
    hipEventRecord(start,0);
    for(int i=0;i<FULL;i+=N)
    {
        //CPU->GPU
        hipMemcpyAsync(a_d,a_h+i,N*sizeof(int),hipMemcpyHostToDevice,stream0);
        hipMemcpyAsync(b_d,b_h+i,N*sizeof(int),hipMemcpyHostToDevice,stream0);
       kernel_add<<<N/256,256,0,stream0>>>(a_d,b_d,c_d);
       //GPU->CPU
       hipMemcpyAsync(c_h+i,c_d,N*sizeof(int),hipMemcpyDeviceToHost,stream0);
    }
    
    hipStreamSynchronize(stream0);
    hipEventRecord(stop);
    //hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    printf("time 1 stream=%fms\n",elapsedTime);
//使用两个流的方法
    hipEventRecord(start,0);
    for(int i=0;i<FULL;i+=2*N)
    {
        hipMemcpyAsync(a_d,a_h+i,N*sizeof(int),hipMemcpyHostToDevice,stream0);           
        hipMemcpyAsync(a1_d,a_h+i+N,N*sizeof(int),hipMemcpyHostToDevice,stream1);
        hipMemcpyAsync(b_d,b_h+i,N*sizeof(int),hipMemcpyHostToDevice,stream0);
        hipMemcpyAsync(b1_d,b_h+i+N,N*sizeof(int),hipMemcpyHostToDevice,stream1);   
        kernel_add<<<N/256,256,0,stream0>>>(a_d,b_d,c_d);
        kernel_add<<<N/256,256,0,stream1>>>(a1_d,b1_d,c1_d);   
        hipMemcpyAsync(c_h+i,c_d,N*sizeof(int),hipMemcpyDeviceToHost,stream0);
        hipMemcpyAsync(c_h+i+N,c1_d,N*sizeof(int),hipMemcpyDeviceToHost,stream1);
    } 
    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);
    hipEventRecord(stop,0);
   // hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);
    printf("time 2 stream=%fms\n",elapsedTime);
    //验证
    for(int i=0;i<FULL;i++)
    {
        if(c_h[i]-a_h[i]-b_h[i]!=0)
        {
            printf("wrong at:%d\n",i);
             exit(EXIT_FAILURE);
        }
        
    }
    printf("pass\n");
    //使用四个流的方法
    // hipEventRecord(start,0);
    // for(int i=0;i<FULL;i+=4*N)
    // {
    //     hipMemcpyAsync(a_d,a_h+i,N*sizeof(int),hipMemcpyHostToDevice,stream0);           
    //     hipMemcpyAsync(a1_d,a_h+i+N,N*sizeof(int),hipMemcpyHostToDevice,stream1);
    //     hipMemcpyAsync(a2_d,a_h+i+2*N,N*sizeof(int),hipMemcpyHostToDevice,stream2);
    //     hipMemcpyAsync(a3_d,a_h+i+3*N,N*sizeof(int),hipMemcpyHostToDevice,stream3);

    //     hipMemcpyAsync(b_d,b_h+i,N*sizeof(int),hipMemcpyHostToDevice,stream0);
    //     hipMemcpyAsync(b1_d,b_h+i+N,N*sizeof(int),hipMemcpyHostToDevice,stream1); 
    //     hipMemcpyAsync(b2_d,b_h+i+2*N,N*sizeof(int),hipMemcpyHostToDevice,stream2);
    //     hipMemcpyAsync(b3_d,b_h+i+3*N,N*sizeof(int),hipMemcpyHostToDevice,stream3);  
       
    //     kernel_add<<<N/256,256,0,stream0>>>(a_d,b_d,c_d);
    //     kernel_add<<<N/256,256,0,stream1>>>(a1_d,b1_d,c1_d);
    //     kernel_add<<<N/256,256,0,stream2>>>(a2_d,b2_d,c2_d); 
    //     kernel_add<<<N/256,256,0,stream3>>>(a3_d,b3_d,c3_d); 

    //     hipMemcpyAsync(c_h+i,c_d,N*sizeof(int),hipMemcpyDeviceToHost,stream0);
    //     hipMemcpyAsync(c_h+i+N,c1_d,N*sizeof(int),hipMemcpyDeviceToHost,stream1);
    //     hipMemcpyAsync(c_h+i+2*N,c_d,N*sizeof(int),hipMemcpyDeviceToHost,stream2);
    //     hipMemcpyAsync(c_h+i+3*N,c_d,N*sizeof(int),hipMemcpyDeviceToHost,stream3);
    // } 
    // hipStreamSynchronize(stream0);
    // hipStreamSynchronize(stream1);
    // hipStreamSynchronize(stream2);
    // hipStreamSynchronize(stream3);
    // hipEventRecord(stop,0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&elapsedTime,start,stop);
    // printf("time 4 stream=%fms\n",elapsedTime);

    //释放
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    hipHostFree(a_h);
    hipHostFree(b_h);
    hipHostFree(c_h);
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    return 0;
} 